// Using CUDA device to calculate pi
#include <stdio.h>
#include <hip/hip_runtime.h>
extern "C" double getTime(void);

#define NBIN 1000000000  // Number of bins
//#define NUM_BLOCK  (300)  // Number of thread blocks
//#define NUM_THREAD  (80)  // Number of threads per block

// Kernel that executes on the CUDA device
__global__ void cal_pi(double *sum, int nbin, double step, int nthreads, int nblocks) {
	int i;
	double x;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
	for (i=idx; i< nbin; i+=nthreads*nblocks) {
		x = (i+0.5)*step;
		sum[idx] += 4.0/(1.0+x*x);
	}
}

// Main routine that executes on the host
int main(void) {
    int num_blocks[8] = {60, 120, 180, 240, 300, 360, 420, 600};
    int num_threads[10] = {16, 32, 48, 64, 80, 96, 112, 128, 144, 160};

    for (int i=0; i < 8; i++) {
        for (int j=0; j < 10; j++) {
            int NUM_BLOCK = num_blocks[i];
            int NUM_THREAD = num_threads[j];

        dim3 dimGrid(NUM_BLOCK,1,1);  // Grid dimensions
        dim3 dimBlock(NUM_THREAD,1,1);  // Block dimensions
        double *sumHost, *sumDev;  // Pointer to host & device arrays
        double pi = 0;
        int tid;

        double step = 1.0/NBIN;  // Step size
        size_t size = NUM_BLOCK*NUM_THREAD*sizeof(double);  //Array memory size
        sumHost = (double *)malloc(size);  //  Allocate array on host
        hipMalloc((void **) &sumDev, size);  // Allocate array on device
        double start = getTime();

        // Initialize array in device to 0
        hipMemset(sumDev, 0, size);
        // Do calculation on device
        cal_pi <<<dimGrid, dimBlock>>> (sumDev, NBIN, step, NUM_THREAD, NUM_BLOCK); // call CUDA kernel
        // Retrieve result from device and store it in host array
        hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
        for(tid=0; tid<NUM_THREAD*NUM_BLOCK; tid++)
            pi += sumHost[tid];
        pi *= step;

        // Print results
        double delta = getTime() - start;
        printf("PI = %.16g computed in %.4g seconds, on NUM_BLOCK: %d, NUM_THREAD: %d\n", pi, delta, NUM_BLOCK, NUM_THREAD);
        // Cleanup
        free(sumHost);
        hipFree(sumDev);

        }
    }

    return 0;
}

