#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCKSIZE 256

__global__ void kern_set_val (float *gpu_ptr, float value, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  //TODO: evaluate the value of i
  if (i < n) gpu_ptr[i] = value;
}

int main () {
  int i, failed=0;
  int N = 1024;   // size of vector
  float value = 11.0; //value ptr should be initialized to
  float *ptr;     // Host pointer 
  float *gpu_ptr; // Device pointer
  
  /* Allocate vector in Host*/
  ptr = (float *)malloc(sizeof(float)*N);
  /* Allocate vector in Device*/
  hipMalloc (&gpu_ptr, sizeof(float)*N);

  //TODO : write kernel invocation here
  //call kernel, takes the value to be initialized
    kern_set_val<<<(N+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(gpu_ptr, value, N);

  hipDeviceSynchronize ();

  //TODO : copy data to host
  hipMemcpy(ptr, gpu_ptr, sizeof(float)*N, hipMemcpyDeviceToHost);

  hipFree (gpu_ptr);

  /* Now check that it did what we want */

  for (i = 0; i < 10; i++)//first ten values are written
    printf ("%f\t", ptr[i]);
  printf ("\n");
  for (i = N-10; i < N; i++)//last ten values are written
    printf ("%f\t", ptr[i]);
  printf ("\n");

  for (i = 0; i < N; i++) {//All values are compared
    if (fabs(ptr[i]-value) > 1e-8) {
      failed=1;
    }
  }
  if (failed) {
    printf ("FAILED !!\n");
  } else {
    printf ("PASSED !!\n");
  }    
  free (ptr);
}
