#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define STOP 0
#define START 1

#define BLOCKSIZE 256 // number of threads per block


extern "C" void chrono (int kind, float *time);

__global__ void kconvol (float *gpu_a, float *gpu_b, int n) {
  int i, j, l;
  // TODO : evaluate the global 1D index l of the current thread,
  // using blockDim, blockIdx and threadIdx.
  l = threadIdx.x + blockIdx.x * blockDim.x;

  // TODO : evaluate global indeces of thread (i,j) from the index l
  j = l/n;
  i = l - j*n;

  if ((i >= n) || (j >= n)) return;
  if ((i == 0) || (j == 0) || (i == n-1) || (j == n-1))  {
    gpu_b[l] = gpu_a[l]; // edges are untouched
  }
  else
    // TODO : fill up the MISSING indices below
    gpu_b[l]=(1./5.)*(gpu_a[l-n] + gpu_a[l-1] + gpu_a[l] + gpu_a[l+1]+ gpu_a[l+n]);
}

extern "C" void gpu_convol (float *a, float *b, int n) {
  float *gpu_a;
  float *gpu_b;
  hipError_t err;
  float time;
  
  err = hipMalloc (&gpu_a, n*n*sizeof(float));
  if (err != 0) {
    printf ("Error allocating gpu_a: %s\n", hipGetErrorString (err));
    exit (1);
  }
  err = hipMalloc (&gpu_b, n*n*sizeof(float));
  if (err != 0) {
    printf ("Error allocating gpu_b: %s\n", hipGetErrorString (err));
    exit (1);
  }
  
  hipMemcpy (gpu_a, a, n*n*sizeof(float), hipMemcpyHostToDevice);
    
  // NOTE : the chronometer below does not contemplate overhead of memory allocation and
  // memory transfer.
  chrono (START, &time);
  // TODO : the number of blocks is missing below in the kernel invocation
  int numBlocks = (n*n + BLOCKSIZE -1) / BLOCKSIZE;
  kconvol <<<numBlocks,BLOCKSIZE>>> (gpu_a, gpu_b, n);
  err=hipDeviceSynchronize ();
  chrono (STOP, &time);
  printf ("Convolution took  %f sec. on GPU\n", time);
  hipMemcpy (b, gpu_b, n*n*sizeof(float), hipMemcpyDeviceToHost);
  if (err != 0) {
    printf ("%s\n", hipGetErrorString (err));
    exit (1);
  }
  hipFree (gpu_a);
  hipFree (gpu_b);
}
